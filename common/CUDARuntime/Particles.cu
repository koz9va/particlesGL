#include "hip/hip_runtime.h"
#include "Particles.cuh"
#include "hip/hip_runtime.h"
#include <>


#define gravity 9.8
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
__global__ void Sand_kernel(point_t *points_d, int *cells_d, int pAm, int w, int h) {

	int i;
	unsigned int id;
	point_t *p;
	bool collided;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < pAm) {
		p = points_d + id;
		switch (p->type) {
			case sand:
				for (i = 1; i < (int)p->speed/2 + 1; ++i) {

					if ((cells_d[ p->x + (p->y + i) * w] != -1) || (p->y + i) >= h - 3) {
						collided = true;
						break;
					}
				}

				if(i > 1 || (i == 1 && !collided)) {
					--i;
					cells_d[p->x + p->y * w ] = -1;
					cells_d[p->x + (p->y + i) * w] = (int)id;
					p->y += i;
					if(!collided && p->speed + p->velocity <= gravity) {
						p->speed += p->velocity;
					} else if(collided)
						p->speed = 1;
					return;
				}


				collided = false;
				for (i = 1; i < (int)p->speed/2 + 1; ++i) {
					if ((cells_d[(p->x + i) + (p->y + i) * w] != -1) || (p->y + i) >= h - 3 || (p->x + i) > w - 3) {
						collided = true;
						break;
					}
				}

				if(i > 1 || (i == 1 && !collided)) {
					--i;
					cells_d[p->x + p->y * w] = -1;
					cells_d[(p->x + i) + (p->y + i) * w] = (int)id;
					p->y += i;
					p->x += i;
					if(!collided && p->speed / 2.0 >= 1.0) {
						p->speed /= 1.25;
					} else
						p->speed = 1;
					return;
				}

				collided = false;
				for (i = 1; i < (int) p->speed/2 + 1; ++i) {
					if ((cells_d[(p->x - i) + (p->y + i) * w] != -1) || (p->y + i) >= h - 3 || (p->x - i) < 3) {
						collided = true;
						break;
					}
				}

				if(i > 1 || (i == 1 && !collided)) {
					--i;
					cells_d[p->x + p->y * w] = -1;
					cells_d[(p->x + i) + (p->y + i) * w] = (int)id;
					p->y += i;
					p->x -= i;
					if(!collided && p->speed / 2.0 <= 1.0) {
						p->speed /= 1.25;
					} else
						p->speed = 1;
					return;
				}
				break;
			case rock:
				return;
			case water:
				return;
		}
	}
}

namespace cu {
	constexpr int blockSize = 256;
	extern "C"{
	void updateSand(point_t *points, int *cells, point_t *points_d, int *cells_d, int pAm, int w, int h, int updated)
		{
		int NumOfBlocks;
		if(updated == 1) {
			hipMemcpy(points_d, points, sizeof(point_t) * pAm, hipMemcpyHostToDevice);
			hipMemcpy(cells_d, cells, sizeof(int) * w * h, hipMemcpyHostToDevice);
		}
		NumOfBlocks = (int)((double)pAm/(double)blockSize) + 1;

		Sand_kernel<<<NumOfBlocks, blockSize>>>(points_d, cells_d, pAm, w, h);

		hipMemcpy(points, points_d, sizeof(point_t) * pAm, hipMemcpyDeviceToHost);
		hipMemcpy(cells, cells_d, sizeof(int) * w * h, hipMemcpyDeviceToHost);
		}
	void AllocPointsAndCells(int w, int h, point_t **points_d, int **cells_d, point_t *points, int *cells, int pAm) {
		hipMalloc(points_d, sizeof(point_t) * w * h);
		hipMalloc(cells_d, sizeof(int) * w * h);

	}
	void FreePointsAndCells(point_t *points_d, int *cells_d) {
		hipFree(points_d);
		hipFree(cells_d);
	}
	}
}