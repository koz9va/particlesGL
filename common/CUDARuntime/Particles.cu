#include "hip/hip_runtime.h"
#include "Particles.cuh"
#include "hip/hip_runtime.h"
#include <>


#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
__global__ void Sand_kernel(point_t *points_d, int *cells_d, int pAm, int w, int h) {

	int id;
	point_t *p;

	id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < pAm) {
		p = points_d + id;
		if(p->y + 2 > h)
			return;

		if(cells_d[w * p->x + (p->y + 1)] == -1) {
			cells_d[w * p->x + p->y] = -1;
			cells_d[w * p->x + (p->y + 1)] = id;
			++p->y;
			return;
		}
		if( (p->x < w && cells_d[w * (p->x + 1) + (p->y + 1)] == -1) ) {
			cells_d[w * p->x + p->y] = -1;
			cells_d[w * (p->x + 1) + (p->y + 1)] = id;
			++p->x;
			++p->y;
			return;
		}
		if(p->x > 0 && cells_d[w * (p->x - 1) + (p->y + 1)] == -1) {
			cells_d[w * p->x + p->y] = -1;
			cells_d[w * (p->x - 1) + (p->y + 1)] = id;
			--p->x;
			++p->y;
			return;
		}

	}
	//@todo implement sand velocity and call this shit from updateSand();
}

namespace cu {
	constexpr int blockSize = 256;
	extern "C"{
	void updateSand(point_t *points, int *cells, point_t *points_d, int *cells_d, int pAm, int w, int h, int updated)
		{
		int NumOfBlocks;
		if(updated == 1) {
			hipMemcpy(points_d, points, sizeof(point_t) * pAm, hipMemcpyHostToDevice);
			hipMemcpy(cells_d, cells, sizeof(int) * w * h, hipMemcpyHostToDevice);
		}
		NumOfBlocks = (int)((double)pAm/(double)blockSize) + 1;

		Sand_kernel<<<NumOfBlocks, blockSize>>>(points_d, cells_d, pAm, w, h);

		hipMemcpy(points, points_d, sizeof(point_t) * pAm, hipMemcpyDeviceToHost);
		hipMemcpy(cells, cells_d, sizeof(int) * w * h, hipMemcpyDeviceToHost);

		}
	void AllocPointsAndCells(int w, int h, point_t **points_d, int **cells_d, point_t *points, int *cells, int pAm) {
		hipMalloc(points_d, sizeof(point_t) * w * h);
		hipMalloc(cells_d, sizeof(int) * w * h);

	}
	void FreePointsAndCells(point_t *points_d, int *cells_d) {
		hipFree(points_d);
		hipFree(cells_d);
	}
	}
}